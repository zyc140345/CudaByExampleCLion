#include "hip/hip_runtime.h"
//
// Created by 张易诚 on 24-9-15.
//

#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define DIM 1000
#define MAX_ITER 100

__device__ int burningShip(int x, int y) {
    const float scale = 1.7;
    float scaled_x = scale * ((float) (x - DIM / 2) / (DIM / 2) - 0.25);
    float scaled_y = scale * ((float) (DIM / 2 - y) / (DIM / 2) - 0.15);

    float zx = scaled_x;
    float zy = scaled_y;

    int i;
    for (i = 0; i < MAX_ITER; i++) {
        float xtemp = zx * zx - zy * zy + scaled_x;
        zy = std::abs(2 * zx * zy) + scaled_y;
        zx = xtemp;
        if (zx * zx + zy * zy >= 4)
            break;
    }

    return i;
}

__device__ float iter_to_magnitude(int iter) {
    return sqrtf(1 - pow(iter / (float) MAX_ITER - 1, 2));
}

__global__ void kernel(unsigned char *ptr) {
    // map from blockIdx to pixel position
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // now calculate the value at that position
    int iter = burningShip(x, y);
    if (iter == MAX_ITER) {
        ptr[offset * 4 + 0] = 0;
        ptr[offset * 4 + 1] = 0;
        ptr[offset * 4 + 2] = 0;
        ptr[offset * 4 + 3] = 255;
    } else {
        auto redValue = (unsigned char) (iter_to_magnitude(iter) * (255 - 82) * 82 / 115);
        auto greenValue = (unsigned char) (iter_to_magnitude(iter) * (255 - 115));
        auto blueValue = (unsigned char) (iter_to_magnitude(iter) * (255 - 37) * 37 / 115);
        ptr[offset * 4 + 0] = 82 + redValue;
        ptr[offset * 4 + 1] = 115 + greenValue;
        ptr[offset * 4 + 2] = 37 + blueValue;
        ptr[offset * 4 + 3] = 255;
    }
}

// globals needed by the update routine
struct DataBlock {
    unsigned char *dev_bitmap;
};

int main(void) {
    DataBlock data;
    CPUBitmap bitmap(DIM, DIM, &data);
    unsigned char *dev_bitmap;

    HANDLE_ERROR(hipMalloc((void **) &dev_bitmap, bitmap.image_size()));
    data.dev_bitmap = dev_bitmap;

    dim3 grid(DIM, DIM);
    kernel<<<grid, 1>>>(dev_bitmap);

    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap,
                            bitmap.image_size(),
                            hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_bitmap));

    bitmap.display_and_exit();
}
